#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>

static void HandleError(hipError_t err, const char *file, int line) { if (err != hipSuccess) { 
	printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);        
	exit(EXIT_FAILURE); } 
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot_kernel(float* a, float* b, float* c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < N)
	{
		temp += a[tid] * b[tid];
		//tid += threadsPerBlock*blocksPerGrid;
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;
	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0)
	{
		if (cacheIndex < i)
			cache[cacheIndex] = cache[cacheIndex] + cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	// ���������ͬ��֮��chache[0]����������Ҫ�ĸ�block�ĺ�
	// �����ֵ����global memory�ֻ��Ҫһ���߳���������£�����ѡ0�߳�
	if (cacheIndex == 0)
	{
		c[blockIdx.x] = cache[0];
	}


}

int main()
{
	float *h_a, *h_b, c, *h_partial_c;
	float *d_a, *d_b, *d_partial_c;

	h_a = (float*)malloc(N * sizeof(float));
	h_b = (float*)malloc(N * sizeof(float));
	h_partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

	HANDLE_ERROR(hipMalloc((void**)&d_a, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_b, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_partial_c, blocksPerGrid * sizeof(float)));

	// fill in data
	for (int i = 0; i < N; i++)
	{
		h_a[i] = i;
		h_b[i] = i * 2;
	}

	HANDLE_ERROR(hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice));

	// launch kernel
	dot_kernel << <threadsPerBlock, blocksPerGrid >> > (d_a, d_b, d_partial_c);

	HANDLE_ERROR(hipMemcpy(h_partial_c, d_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
	{
		c += h_partial_c[i];
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_partial_c);
	free(h_a);
	free(h_b);
	free(h_partial_c);

	printf("Square Sum:%f\n", c);
}