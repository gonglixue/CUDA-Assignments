#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)  // �ֳ�20��chunk?

__global__ void kernel(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N)
	{
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main()
{
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap) {
		printf("Device will not not handle overlaps, so no speed up from streams\n");
		return 0;
	}

	hipEvent_t start, stop;
	float elapsedTime;
	// start the timers
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// create a stream
	hipStream_t stream;
	hipStreamCreate(&stream);

	// prepare data
	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	// loop over full data, in bite-sized chunks
	for (int i = 0; i < FULL_DATA_SIZE; i += N)
	{
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		// ��Щ�����첽�ģ�a���ص�ʱ��δ��ִ���꣬����һ�ζ�������д���ʱ��aһ��ִ���ꡣ
		// Ҳ���ǣ����￪ʼд��b��ʱ��a�ѿ����ꡣ
		// kernelִ��ǰ��b�Ѿ������ꡣ
		kernel << <N / 256, 256, 0, stream >> > (dev_a, dev_b, dev_c);

		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}
	// ��forѭ��ִ���꣬GPU���ܻ����������δ���
	hipStreamSynchronize(stream);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,
		start, stop);
	printf("Time taken: %3.1f ms\n", elapsedTime);
	// cleanup the streams and memory
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	hipStreamDestroy(stream);
	return 0;
}